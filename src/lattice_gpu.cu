#include "hip/hip_runtime.h"
/**
 * @file
 *
 * @author Alessandro Renna <alessandro1.renna@mail.polimi.it>
 * @author Mattia Marzotto <mattia.marzotto@mail.polimi.it>
 */

#include "src/lattice_gpu.cuh"

// __global__ void hello(void)
// {
//   printf("Hello from block %d, thread %d\n", blockIdx.x, threadIdx.x);
// }

__device__ double compute_equilibrium(const double *d_weights, const double *d_coeff,
                                      double rho, double ux, double uy, int i)
{
  double weight = d_weights[i];
  double cx = d_coeff[2 * i];
  double cy = d_coeff[2 * i + 1];
  double u_dot_c = ux * cx + uy * cy;
  double u_sq = ux * ux + uy * uy;
  return weight * rho * (1.0 + 3.0 * u_dot_c + 4.5 * u_dot_c * u_dot_c - 1.5 * u_sq);
}

__device__ int find_forward_index(int current_index, int nx, int ny, int i, const double *d_coeff)
{
  int x = current_index % nx;
  int y = current_index / nx;
  double cx = d_coeff[2 * i];
  double cy = d_coeff[2 * i + 1];
  int x_new = x + cx;
  int y_new = y + cy;
  return y_new * nx + x_new;
}

__device__ int find_backward_index(int current_index, int nx, int ny, int i, const double *d_coeff, const int *d_bb_indexes)
{
  int x = current_index % nx;
  int y = current_index / nx;
  double cx = d_coeff[2 * d_bb_indexes[i]];
  double cy = d_coeff[2 * d_bb_indexes[i] + 1];
  int x_new = x + cx;
  int y_new = y + cy;
  return y_new * nx + x_new;
}

__device__ bool check_backward(int index, int nx, int ny, int i, const double *d_coeff, const int *d_bb_indexes, NodeType *d_node_types)
{
  int backward_index = find_backward_index(index, nx, ny, i, d_coeff, d_bb_indexes);
  return d_node_types[backward_index] == NodeType::fluid || d_node_types[backward_index] == NodeType::boundary;
}

__device__ void apply_IBB(const int dir,const double *d_weights, const double *d_coeff, const int *d_bb_indexes, 
                          double *d_f_post, double *d_f_adj,
                          double *d_ux, double *d_uy, double *d_rho,
                          NodeType *d_node_types, bool * d_bounce_back_dir, double * d_bounce_back_delta,
                          int nx, int ny, int i, int index)
{
  int forward_index = find_forward_index(index, nx, ny, i, d_coeff);
  double ux_wall = d_ux[forward_index];
  double uy_wall = d_uy[forward_index];

  if(check_backward(index, nx, ny, i, d_coeff, d_bb_indexes, d_node_types))
  {
    double cx = d_coeff[2 * i];
    double cy = d_coeff[2 * i + 1];

    double f_adj_post_coll = d_f_adj[index * dir + i];
    d_f_adj[index * dir + d_bb_indexes[i]] = (2 * d_bounce_back_delta[index * dir + i] * d_f_post[index * dir + i] + 
                    (1 - 2 * d_bounce_back_delta[index * dir + i]) * f_adj_post_coll) * 
                    (d_bounce_back_delta[index * dir + i] < 0.5) +
                    (1. / (2 * d_bounce_back_delta[index * dir + i]) * d_f_post[index * dir + i] + 
                    ((2 * d_bounce_back_delta[index * dir + i] - 1.) / (2 * d_bounce_back_delta[index * dir + i])) * d_f_post[index * dir + d_bb_indexes[i]]) *
                    (d_bounce_back_delta[index * dir + i] >= 0.5) - 
                    (ux_wall * cx + uy_wall * cy) * d_weights[i] * 6;
  }
  else
  {
    d_f_adj[index * dir + d_bb_indexes[i]] = d_f_post[index * dir + i];
  }
}

__device__ void apply_anti_BB(const int dir,const double *d_weights, const double *d_coeff, const int *d_bb_indexes, 
                          double *d_f_post, double *d_f_adj,
                          double *d_ux, double *d_uy, double *d_rho,
                          NodeType *d_node_types, bool * d_bounce_back_dir, double * d_bounce_back_delta,
                          int nx, int ny, int i, int index)
{
  int forward_index = find_forward_index(index, nx, ny, i, d_coeff);
  double ux_wall = d_ux[forward_index];
  double uy_wall = d_uy[forward_index];
  double rho_wall = d_rho[forward_index];

  if(d_node_types[forward_index] == NodeType::outlet)
  {
    int backward_index = find_backward_index(index, nx, ny, i, d_coeff, d_bb_indexes);
    double ux_fluid = d_ux[backward_index];
    double uy_fluid = d_uy[backward_index];

    ux_wall = (d_ux[forward_index] + ux_fluid) / 2;
    uy_wall = (d_uy[forward_index] + uy_fluid) / 2;
    
    rho_wall = 0.8 * (2 * (d_f_post[index * dir + 1] + d_f_post[index * dir + 5] + d_f_post[index * dir + 8]) + d_f_post[index * dir + 0] + d_f_post[index * dir + 2] + d_f_post[index * dir + 4]) / (1. - ux_wall);
  }

  if(check_backward(index, nx, ny, i, d_coeff, d_bb_indexes, d_node_types))
  {
    double cx = d_coeff[2 * i];
    double cy = d_coeff[2 * i + 1];

    d_f_adj[index * dir + d_bb_indexes[i]] = -d_f_post[index * dir + i] +
                          2 * d_weights[i] * rho_wall *
                          (1 + 4.5 * (cx * ux_wall + cy * uy_wall) * (cx * ux_wall + cy * uy_wall) -
                          3.5 * (ux_wall * ux_wall + uy_wall * uy_wall));
  }
  else
  {
    d_f_adj[index * dir + d_bb_indexes[i]] = d_f_post[index * dir + i];
  }
}

__global__ void collide_and_stream_kernel(
  const int dir, const double *d_weights, const double *d_coeff, const int *d_bb_indexes,
  double *d_f_pre, double *d_f_post, double *d_f_adj, 
  double *d_ux, double *d_uy, double *d_rho, 
  NodeType *d_node_types, bool * d_bounce_back_dir, int nx, int ny, double tau) 
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int n = nx * ny;

  if(index < n) 
  {
    if(d_node_types[index] == NodeType::fluid || d_node_types[index] == NodeType::boundary) 
    {
      // Collision step
      for(int i = 0; i < dir; ++i)
      {
        double feq = compute_equilibrium(d_weights, d_coeff, d_rho[index], d_ux[index], d_uy[index], i);
        d_f_post[index * dir + i] = d_f_pre[index * dir + i] - (d_f_pre[index * dir + i] - feq) / tau;
      }

      // Streaming step
      d_f_adj[index * dir + 0] = d_f_post[index * dir + 0];
      for(int i = 0; i < dir; ++i) 
      {
        if(!d_bounce_back_dir[index * dir + i]) {
          int index_new = find_forward_index(index, nx, ny, i, d_coeff);
          d_f_adj[index_new * dir + i] = d_f_post[index * dir + i];
        }
      }
    }
  }
}

__global__ void apply_BCs_and_compute_quantities_kernel(
  const int dir, const double *d_weights, const double *d_coeff, const int *d_bb_indexes,
  double *d_f_pre, double *d_f_post, double *d_f_adj,
  double *d_ux, double *d_uy, double *d_rho,
  double *d_drag, double *d_lift, bool * d_obstacle,
  NodeType *d_node_types, bool * d_bounce_back_dir, double * d_bounce_back_delta, int nx, int ny) 
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int n = nx * ny;

  if(index < n)
  {
    if(d_node_types[index] == NodeType::fluid || d_node_types[index] == NodeType::boundary)
    {
      if(d_node_types[index] == NodeType::boundary)
      {
        // Apply boundary conditions
        for(int i = 0; i < dir; ++i)
        {
          if(d_bounce_back_dir[index * dir + i])
          {
            int index_new = find_forward_index(index, nx, ny, i, d_coeff);

            if(d_node_types[index_new] == NodeType::solid ||
              d_node_types[index_new] == NodeType::obstacle ||
              d_node_types[index_new] == NodeType::inlet)
            {
              // Interpolated Bounce-Back
              apply_IBB(dir, d_weights, d_coeff, d_bb_indexes, d_f_post, d_f_adj, d_ux, d_uy, d_rho, d_node_types, d_bounce_back_dir, d_bounce_back_delta, nx, ny, i, index);
            }
            else if(d_node_types[index_new] == NodeType::outlet)
            {
              // Anti Bounce-Back
              apply_anti_BB(dir, d_weights, d_coeff, d_bb_indexes, d_f_post, d_f_adj, d_ux, d_uy, d_rho, d_node_types, d_bounce_back_dir, d_bounce_back_delta, nx, ny, i, index);
            }
            else
            {
              printf("Error: Invalid BCs type at index %d\n", index);
              return;
            }
          }
        }

        // Compute drag and lift
        if(*d_obstacle)
        {
          double dr = 0.0;
          double lf = 0.0;
          for(int i = 0; i < dir; ++i)
          {
            int forward_index = find_forward_index(index, nx, ny, i, d_coeff);
            if(d_node_types[forward_index] == NodeType::obstacle)
            {
              double cx = d_coeff[2 * i];
              double cy = d_coeff[2 * i + 1];
              double cx_bb = d_coeff[2 * d_bb_indexes[i]];
              double cy_bb = d_coeff[2 * d_bb_indexes[i] + 1];

              dr += cx * d_f_pre[index * dir + i] - cx_bb * d_f_adj[index * dir + d_bb_indexes[i]];
              lf += cy * d_f_pre[index * dir + i] - cy_bb * d_f_adj[index * dir + d_bb_indexes[i]];
            }
          }
          atomicAdd(d_drag, dr);
          atomicAdd(d_lift, lf);
        }
      }

      // Update f
      for(int i = 0; i < dir; ++i)
      {
        d_f_pre[index * dir + i] = d_f_adj[index * dir + i];
      }

      // Compute macroscopic quantities
      double rho = 0.0;
      double ux = 0.0;
      double uy = 0.0;
      for(int i = 0; i < dir; ++i)
      {
        double f = d_f_pre[index * dir + i];
        rho += f;
        ux += f * d_coeff[2 * i];
        uy += f * d_coeff[2 * i + 1];
      }
      d_rho[index] = rho;
      d_ux[index] = ux / rho;
      d_uy[index] = uy / rho;
    }
  }
}

void
lbm_gpu::cuda_simulation(unsigned int nx, 
                        unsigned int ny, 
                        std::vector<Node> &nodes,
                        double tau,
                        double dt,
                        unsigned int save_iter,
                        unsigned int max_iter)
{
  const int n = nx * ny;

  // Constants for CUDA kernel
  const int dir = Node::dir;
  const double *weights = vectorToArray(Node::weights);
  const double *coeff = vector2DToArray(Node::coeff);
  const int *bb_indexes = vectorToArray(Node::bb_indexes);

  // Copy constant variables to device
  double *d_weights, *d_coeff;
  int *d_bb_indexes;

  hipMalloc((void **) &d_weights, dir * sizeof(double));
  hipMalloc((void **) &d_coeff, 2 * dir * sizeof(double));
  hipMalloc((void **) &d_bb_indexes, dir * sizeof(int));

  hipMemcpy(d_weights, weights, dir * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_coeff, coeff, 2 * dir * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_bb_indexes, bb_indexes, dir * sizeof(int), hipMemcpyHostToDevice);


  std::vector<double> lift_out(max_iter, 0.0);
  std::vector<double> drag_out(max_iter, 0.0);
  
  // Host variables
  double * host_f_pre, * host_f_post, * host_f_adj, * host_ux, * host_uy, * host_rho, * host_drag, * host_lift, * host_bounce_back_delta;
  int * host_coord;
  bool * host_bounce_back_dir, * host_obstacle;
  NodeType * host_node_types;

  // Device variables
  double * d_f_pre, * d_f_post, * d_f_adj, * d_ux, * d_uy, * d_rho, * d_drag, * d_lift, * d_bounce_back_delta;
  int * d_coord;
  bool * d_bounce_back_dir, * d_obstacle;
  NodeType * d_node_types;

  // Allocate memory on the host
  hipHostMalloc((void **) &host_f_pre, n * dir * sizeof(double));
  hipHostMalloc((void **) &host_f_post, n * dir * sizeof(double));
  hipHostMalloc((void **) &host_f_adj, n * dir * sizeof(double));
  hipHostMalloc((void **) &host_ux, n * sizeof(double));
  hipHostMalloc((void **) &host_uy, n * sizeof(double));
  hipHostMalloc((void **) &host_rho, n * sizeof(double));
  hipHostMalloc((void **) &host_drag, sizeof(double));
  hipHostMalloc((void **) &host_lift, sizeof(double));
  hipHostMalloc((void **) &host_coord, n * 2 * sizeof(int));
  hipHostMalloc((void **) &host_bounce_back_delta, n * dir * sizeof(double));
  hipHostMalloc((void **) &host_bounce_back_dir, n * dir * sizeof(bool));
  hipHostMalloc((void **) &host_obstacle, sizeof(bool));
  hipHostMalloc((void **) &host_node_types, n * sizeof(NodeType));

  // Initialize host_obstacle
  *host_obstacle = false;

  // Allocate memory on the device
  hipMalloc((void **) &d_f_pre, n * dir * sizeof(double));
  hipMalloc((void **) &d_f_post, n * dir * sizeof(double));
  hipMalloc((void **) &d_f_adj, n * dir * sizeof(double));
  hipMalloc((void **) &d_ux, n * sizeof(double));
  hipMalloc((void **) &d_uy, n * sizeof(double));
  hipMalloc((void **) &d_rho, n * sizeof(double));
  hipMalloc((void **) &d_drag, sizeof(double));
  hipMalloc((void **) &d_lift, sizeof(double));
  hipMalloc((void **) &d_coord, n * 2 * sizeof(int));
  hipMalloc((void **) &d_bounce_back_delta, n * dir * sizeof(double));
  hipMalloc((void **) &d_bounce_back_dir, n * dir * sizeof(bool));
  hipMalloc((void **) &d_obstacle, sizeof(bool));
  hipMalloc((void **) &d_node_types, n * sizeof(NodeType));

  // Set host data
  for(unsigned int index = 0; index < n; index++)
  {
    std::vector<double> temp_f_pre = nodes[index].get_f_pre();
    std::vector<double> temp_f_post = nodes[index].get_f_post();
    std::vector<double> temp_f_adj = nodes[index].get_f_adj();
    std::vector<double> temp_bounce_back_delta = nodes[index].get_bounce_back_delta();
    std::vector<bool> temp_bounce_back_dir = nodes[index].get_bounce_back_dir();

    host_ux[index] = nodes[index].get_ux();
    host_uy[index] = nodes[index].get_uy();
    host_rho[index] = nodes[index].get_rho();
    host_node_types[index] = nodes[index].get_node_type();
    
    if(host_node_types[index] == NodeType::obstacle && !(*host_obstacle)) {
      *host_obstacle = true;
    }

    for(unsigned int i = 0; i < dir; i++)
    {
      host_f_pre[index * dir + i] = temp_f_pre[i];
      host_f_post[index * dir + i] = temp_f_post[i];
      host_f_adj[index * dir + i] = temp_f_adj[i];
      host_bounce_back_delta[index * dir + i] = temp_bounce_back_delta[i];
      host_bounce_back_dir[index * dir + i] = temp_bounce_back_dir[i];
    }

    for(unsigned int i = 0; i < 2; i++)
    {
      host_coord[index * 2 + i] = nodes[index].get_coord()[i];
    }
    
  }
  *host_drag = 0.0;
  *host_lift = 0.0;

  std::cout << "Copying data to device\n" << std::endl;
  // Copy data from host to device
  hipMemcpy(d_f_pre, host_f_pre, n * dir * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_f_post, host_f_post, n * dir * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_f_adj, host_f_adj, n * dir * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_ux, host_ux, n * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_uy, host_uy, n * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_rho, host_rho, n * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_drag, host_drag, sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_lift, host_lift, sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_coord, host_coord, n * 2 * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_bounce_back_delta, host_bounce_back_delta, n * dir * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_bounce_back_dir, host_bounce_back_dir, n * dir * sizeof(bool), hipMemcpyHostToDevice);
  hipMemcpy(d_obstacle, host_obstacle, sizeof(bool), hipMemcpyHostToDevice);
  hipMemcpy(d_node_types, host_node_types, n * sizeof(NodeType), hipMemcpyHostToDevice);

  // Free host memory
  hipHostFree(host_f_pre);
  hipHostFree(host_f_post);
  hipHostFree(host_f_adj);
  hipHostFree(host_coord);
  hipHostFree(host_bounce_back_delta);
  hipHostFree(host_bounce_back_dir);
  hipHostFree(host_obstacle);
  hipHostFree(host_node_types);


  // Run simulation
  std::cout << "Running simulation\n" << std::endl;
  auto start_time = std::chrono::high_resolution_clock::now();
  unsigned int iter = 0;
  double total_time = 0.0;
  std::cout << "Create folder and files\n" << std::endl;
  // Delete the output_results directory if it exists
  if (std::filesystem::exists("output_results")) {
    std::filesystem::remove_all("output_results");
    std::filesystem::create_directory("output_results");
  }
  else{
    std::filesystem::create_directory("output_results");
  }

  if (std::filesystem::exists("output_animations")) {
    std::filesystem::remove_all("output_animations");
    std::filesystem::create_directory("output_animations");
  }
  else{
    std::filesystem::create_directory("output_animations");
  }
  
  std::string u_filename = "output_results/velocity_out.txt";
  std::string ux_filename = "output_results/ux_out.txt";
  std::string uy_filename = "output_results/uy_out.txt";
  std::string rho_filename = "output_results/rho_out.txt";

  std::ofstream u_file(u_filename);
  std::ofstream ux_file(ux_filename);
  std::ofstream uy_file(uy_filename);
  std::ofstream rho_file(rho_filename);

  std::cout << "Save initial conditions\n" << std::endl;
  std::vector<double> vec_ux(nx * ny), vec_uy(nx * ny), vec_rho(nx * ny);
  vec_ux = arrayToVector(host_ux, nx * ny);
  vec_uy = arrayToVector(host_uy, nx * ny);
  vec_rho = arrayToVector(host_rho, nx * ny);
  // Save the initial conditions
  writeResults(u_file, ux_file, uy_file, rho_file, vec_ux, vec_uy, vec_rho, nx, ny);

  iter = iter + 1;

  std::cout << "Start simulation loop\n" << std::endl;
  while(iter <= max_iter) {
    auto iter_start_time = std::chrono::high_resolution_clock::now();

    // if(iter % save_iter == 0 || iter == max_iter - 1) {
      std::cout << "Iteration: " << iter << std::endl;
      std::cout << "Time: " << iter * dt << std::endl;
      std::cout << "Collision and streaming" << std::endl;
    // }

    // Define block size
    int blockSize = 256; // 256 or 512

    // Calculate grid size
    int gridSize = (nx * ny + blockSize - 1) / blockSize;

    // Launch CUDA kernel for collision and streaming
    collide_and_stream_kernel<<<gridSize, blockSize>>>(dir, d_weights, d_coeff, d_bb_indexes, 
                                                      d_f_pre, d_f_post, d_f_adj,
                                                      d_ux, d_uy, d_rho, 
                                                      d_node_types, d_bounce_back_dir, nx, ny, tau);

    // if(iter % save_iter == 0 || iter == max_iter - 1) {
      std::cout << "Physical quantities evaluation\n" << std::endl;
    // }

    // Launch CUDA kernel for applying boundary conditions and computing physical quantities
    apply_BCs_and_compute_quantities_kernel<<<gridSize, blockSize>>>(dir, d_weights, d_coeff, d_bb_indexes, 
                                                                    d_f_pre, d_f_post, d_f_adj,
                                                                    d_ux, d_uy, d_rho,
                                                                    d_drag, d_lift, d_obstacle,
                                                                    d_node_types, d_bounce_back_dir, d_bounce_back_delta, nx, ny);

    // Copy lift and drag results from device to host
    hipMemcpy(&host_lift, d_lift, sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(&host_drag, d_drag, sizeof(double), hipMemcpyDeviceToHost);

    lift_out[iter] = *host_lift;
    drag_out[iter] = *host_drag;

    if(iter % save_iter == 0 || iter == max_iter - 1) {
      // Copy results from device to host
      hipMemcpy(host_ux, d_ux, n * sizeof(double), hipMemcpyDeviceToHost);
      hipMemcpy(host_uy, d_uy, n * sizeof(double), hipMemcpyDeviceToHost);
      hipMemcpy(host_rho, d_rho, n * sizeof(double), hipMemcpyDeviceToHost);

      vec_ux = arrayToVector(host_ux, nx * ny);
      vec_uy = arrayToVector(host_uy, nx * ny);
      vec_rho = arrayToVector(host_rho, nx * ny);

      writeResults(u_file, ux_file, uy_file, rho_file, vec_ux, vec_uy, vec_rho, nx, ny);
    }

    iter = iter + 1;
    auto iter_end_time = std::chrono::high_resolution_clock::now();
    total_time += std::chrono::duration<double>(iter_end_time - iter_start_time).count();
  }

  // Save the lift and drag results
  if(host_obstacle)
  { 
    std::string lift_drag_filename = "output_results/lift_&_drag.txt";
    std::ofstream lift_drag_file(lift_drag_filename);
    lift_drag_file << "Lift:\n";
    // Save the lift and drag
    for(unsigned int t=0; t<max_iter; ++t){
      lift_drag_file << lift_out[t] << " ";
    }
    
    lift_drag_file << "\nDrag:\n";
    // Save the lift and drag
    for(unsigned int t=0; t<max_iter; ++t){
      lift_drag_file << drag_out[t] << " ";
    }
    lift_drag_file.close();
  }

  auto end_time = std::chrono::high_resolution_clock::now();
  std::chrono::duration<double> elapsed_time = end_time - start_time;
  double mean_time_per_iter = total_time / max_iter;
  std::cout << "Simulation completed in " << elapsed_time.count() << " seconds" << std::endl;
  std::cout << "Mean time per iteration: " << mean_time_per_iter << " seconds.\n" << std::endl;

  u_file.close();
  ux_file.close();
  uy_file.close();
  rho_file.close();

  // Free device memory
  hipHostFree(host_ux);
  hipHostFree(host_uy);
  hipHostFree(host_rho);
  hipHostFree(host_drag);
  hipHostFree(host_lift);
}