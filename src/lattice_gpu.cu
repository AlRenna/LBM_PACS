#include "hip/hip_runtime.h"
/**
 * @file
 *
 * @author Alessandro Renna <alessandro1.renna@mail.polimi.it>
 * @author Mattia Marzotto <mattia.marzotto@mail.polimi.it>
 */

#include "src/lattice_gpu.cuh"

// __global__ void hello(void)
// {
//   printf("Hello from block %d, thread %d\n", blockIdx.x, threadIdx.x);
// }



void
lbm_gpu::cuda_simulation(unsigned int nx, 
                        unsigned int ny, 
                        std::vector<Node> &nodes,
                        double tau,
                        double dt,
                        unsigned int save_iter,
                        unsigned int max_iter)
{
  const int n = nx * ny;

  const int dir = Node::dir;

  //TODO: controlla per drag e lift se conviene fare in maniera diversa (somma in un unica variabile)

  // Host variables
  double * host_f_pre, * host_f_post, * host_f_adj, * host_ux, * host_uy, * host_rho, * host_drag, * host_lift, * host_bounce_back_delta;
  int * host_coord;
  bool * host_bounce_back_dir, * host_obstacle;
  NodeType * host_node_types;


  // Device variables
  double * d_f_pre, * d_f_post, * d_f_adj, * d_ux, * d_uy, * d_rho, * d_drag, * d_lift, * d_bounce_back_delta;
  int * d_coord;
  bool * d_bounce_back_dir, * d_obstacle;
  NodeType * d_node_types;

  // Allocate memory on the host
  hipMalloc((void **) &host_f_pre, n * dir * sizeof(double));
  hipMalloc((void **) &host_f_post, n * dir * sizeof(double));
  hipMalloc((void **) &host_f_adj, n * dir * sizeof(double));
  hipMalloc((void **) &host_ux, n * sizeof(double));
  hipMalloc((void **) &host_uy, n * sizeof(double));
  hipMalloc((void **) &host_rho, n * sizeof(double));
  hipMalloc((void **) &host_drag, n * sizeof(double));
  hipMalloc((void **) &host_lift, n * sizeof(double));
  hipMalloc((void **) &host_coord, n * 2 * sizeof(int));
  hipMalloc((void **) &host_bounce_back_delta, n * dir * sizeof(double));
  hipMalloc((void **) &host_bounce_back_dir, n * dir * sizeof(bool));
  hipMalloc((void **) &host_obstacle, sizeof(bool));
  hipMalloc((void **) &host_node_types, n * sizeof(NodeType));

  // Allocate memory on the device
  hipMalloc((void **) &d_f_pre, n * dir * sizeof(double));
  hipMalloc((void **) &d_f_post, n * dir * sizeof(double));
  hipMalloc((void **) &d_f_adj, n * dir * sizeof(double));
  hipMalloc((void **) &d_ux, n * sizeof(double));
  hipMalloc((void **) &d_uy, n * sizeof(double));
  hipMalloc((void **) &d_rho, n * sizeof(double));
  hipMalloc((void **) &d_drag, n * sizeof(double));
  hipMalloc((void **) &d_lift, n * sizeof(double));
  hipMalloc((void **) &d_coord, n * 2 * sizeof(int));
  hipMalloc((void **) &d_bounce_back_delta, n * dir * sizeof(double));
  hipMalloc((void **) &d_bounce_back_dir, n * dir * sizeof(bool));
  hipMalloc((void **) &d_obstacle, sizeof(bool));
  hipMalloc((void **) &d_node_types, n * sizeof(NodeType));


  // Set host data
  for(unsigned int index = 0; index < n; index++)
  {
    std::vector<double> temp_f_pre = nodes[index].get_f_pre();
    std::vector<double> temp_f_post = nodes[index].get_f_post();
    std::vector<double> temp_f_adj = nodes[index].get_f_adj();
    std::vector<double> temp_bounce_back_delta = nodes[index].get_bounce_back_delta();
    std::vector<bool> temp_bounce_back_dir = nodes[index].get_bounce_back_dir();

    host_ux[index] = nodes[index].get_ux();
    host_uy[index] = nodes[index].get_uy();
    host_rho[index] = nodes[index].get_rho();
    host_drag[index] = 0.0;
    host_lift[index] = 0.0;
    host_node_types[index] = nodes[index].get_node_type();
    
    if(host_node_types[index] == NodeType::obstacle && !(*host_obstacle)) {
      *host_obstacle = true;
    }

    for(unsigned int i = 0; i < dir; i++)
    {
      host_f_pre[index * dir + i] = temp_f_pre[i];
      host_f_post[index * dir + i] = temp_f_post[i];
      host_f_adj[index * dir + i] = temp_f_adj[i];
      host_bounce_back_delta[index * dir + i] = temp_bounce_back_delta[i];
      host_bounce_back_dir[index * dir + i] = temp_bounce_back_dir[i];
    }

    for(unsigned int i = 0; i < 2; i++)
    {
      host_coord[index * 2 + i] = nodes[index].get_coord()[i];
    }
    
  }

  // Copy data from host to device
  hipMemcpy(d_f_pre, host_f_pre, n * dir * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_f_post, host_f_post, n * dir * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_f_adj, host_f_adj, n * dir * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_ux, host_ux, n * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_uy, host_uy, n * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_rho, host_rho, n * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_drag, host_drag, n * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_lift, host_lift, n * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_coord, host_coord, n * 2 * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_bounce_back_delta, host_bounce_back_delta, n * dir * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_bounce_back_dir, host_bounce_back_dir, n * dir * sizeof(bool), hipMemcpyHostToDevice);
  hipMemcpy(d_obstacle, host_obstacle, sizeof(bool), hipMemcpyHostToDevice);
  hipMemcpy(d_node_types, host_node_types, n * sizeof(NodeType), hipMemcpyHostToDevice);

  // Free host memory
  // TODO: free host memory


  // Run simulation
  std::cout << "Running simulation\n" << std::endl;
  auto start_time = std::chrono::high_resolution_clock::now();
  unsigned int iter = 0;
  double total_time = 0.0;
  // Delete the output_results directory if it exists
  if (std::filesystem::exists("output_results")) {
    std::filesystem::remove_all("output_results");
    std::filesystem::create_directory("output_results");
  }
  else{
    std::filesystem::create_directory("output_results");
  }

  if (std::filesystem::exists("output_animations")) {
    std::filesystem::remove_all("output_animations");
    std::filesystem::create_directory("output_animations");
  }
  else{
    std::filesystem::create_directory("output_animations");
  }
  
  std::string u_filename = "output_results/velocity_out.txt";
  std::string ux_filename = "output_results/ux_out.txt";
  std::string uy_filename = "output_results/uy_out.txt";
  std::string rho_filename = "output_results/rho_out.txt";

  std::ofstream u_file(u_filename);
  std::ofstream ux_file(ux_filename);
  std::ofstream uy_file(uy_filename);
  std::ofstream rho_file(rho_filename);

  
  std::vector<double> vec_ux(nx * ny), vec_uy(nx * ny), vec_rho(nx * ny);
  vec_ux = arrayToVector(host_ux, nx * ny);
  vec_uy = arrayToVector(host_uy, nx * ny);
  vec_rho = arrayToVector(host_rho, nx * ny);
  // Save the initial conditions
  writeResults(u_file, ux_file, uy_file, rho_file, vec_ux, vec_uy, vec_rho, nx, ny);
}