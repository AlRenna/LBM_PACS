#include "hip/hip_runtime.h"
/**
 * @file
 *
 * @author Alessandro Renna <alessandro1.renna@mail.polimi.it>
 * @author Mattia Marzotto <mattia.marzotto@mail.polimi.it>
 */

#include "lattice_gpu.cuh"

__global__ void hello(void)
{
  printf("Hello from block %d, thread %d\n", blockIdx.x, threadIdx.x);
}

void
lbm_gpu::cuda_simulation()
{
  std::cout << "Hello from CUDA!" << std::endl;
  hello <<<1, 1>>> ();
  hipDeviceSynchronize(); // Ensure kernel execution is completed
}