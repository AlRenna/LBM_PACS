#include "hip/hip_runtime.h"
/**
 * @file
 *
 * @author Alessandro Renna <alessandro1.renna@mail.polimi.it>
 * @author Mattia Marzotto <mattia.marzotto@mail.polimi.it>
 */

#include "src/lattice_gpu.cuh"

// __global__ void hello(void)
// {
//   printf("Hello from block %d, thread %d\n", blockIdx.x, threadIdx.x);
// }

#define CUDA_CHECK(call) \
  do { \
    hipError_t err = call; \
    if (err != hipSuccess) { \
      std::cerr << "CUDA error in " << __FILE__ << " at line " << __LINE__ << ": " \
                << hipGetErrorString(err) << std::endl; \
      exit(EXIT_FAILURE); \
    } \
  } while (0)


__device__ double compute_equilibrium(const double *d_weights, const double *d_coeff,
                                      double rho, double ux, double uy, int i)
{
  double weight = d_weights[i];
  double cx = d_coeff[2 * i];
  double cy = d_coeff[2 * i + 1];
  double u_dot_c = ux * cx + uy * cy;
  double u_sq = ux * ux + uy * uy;
  return weight * rho * (1.0 + 3.0 * u_dot_c + 4.5 * u_dot_c * u_dot_c - 1.5 * u_sq);
}

__device__ int find_forward_index(int current_index, int nx, int ny, int i, const double *d_coeff)
{
  int x = current_index % nx;
  int y = current_index / nx;
  double cx = d_coeff[2 * i];
  double cy = d_coeff[2 * i + 1];
  int x_new = x + cx;
  int y_new = y + cy;
  return y_new * nx + x_new;
}

__device__ int find_backward_index(int current_index, int nx, int ny, int i, const double *d_coeff, const int *d_bb_indexes)
{
  int x = current_index % nx;
  int y = current_index / nx;
  double cx = d_coeff[2 * d_bb_indexes[i]];
  double cy = d_coeff[2 * d_bb_indexes[i] + 1];
  int x_new = x + cx;
  int y_new = y + cy;
  return y_new * nx + x_new;
}

__device__ bool check_backward(int index, int nx, int ny, int i, const double *d_coeff, const int *d_bb_indexes, NodeType *d_node_types)
{
  int backward_index = find_backward_index(index, nx, ny, i, d_coeff, d_bb_indexes);
  return d_node_types[backward_index] == NodeType::fluid || d_node_types[backward_index] == NodeType::boundary;
}

__device__ void apply_IBB(const int dir,const double *d_weights, const double *d_coeff, const int *d_bb_indexes, 
                          double *d_f_post, double *d_f_adj,
                          double *d_ux, double *d_uy, double *d_rho,
                          NodeType *d_node_types, bool * d_bounce_back_dir, double * d_bounce_back_delta,
                          int nx, int ny, int i, int index)
{
  int forward_index = find_forward_index(index, nx, ny, i, d_coeff);
  double ux_wall = d_ux[forward_index];
  double uy_wall = d_uy[forward_index];

  if(check_backward(index, nx, ny, i, d_coeff, d_bb_indexes, d_node_types))
  {
    double cx = d_coeff[2 * i];
    double cy = d_coeff[2 * i + 1];

    double f_adj_post_coll = d_f_adj[index * dir + i];
    d_f_adj[index * dir + d_bb_indexes[i]] = (2 * d_bounce_back_delta[index * dir + i] * d_f_post[index * dir + i] + 
                    (1 - 2 * d_bounce_back_delta[index * dir + i]) * f_adj_post_coll) * 
                    (d_bounce_back_delta[index * dir + i] < 0.5) +
                    (1. / (2 * d_bounce_back_delta[index * dir + i]) * d_f_post[index * dir + i] + 
                    ((2 * d_bounce_back_delta[index * dir + i] - 1.) / (2 * d_bounce_back_delta[index * dir + i])) * d_f_post[index * dir + d_bb_indexes[i]]) *
                    (d_bounce_back_delta[index * dir + i] >= 0.5) - 
                    (ux_wall * cx + uy_wall * cy) * d_weights[i] * 6;
  }
  else
  {
    d_f_adj[index * dir + d_bb_indexes[i]] = d_f_post[index * dir + i];
  }
}

__device__ void apply_anti_BB(const int dir,const double *d_weights, const double *d_coeff, const int *d_bb_indexes, 
                          double *d_f_post, double *d_f_adj,
                          double *d_ux, double *d_uy, double *d_rho,
                          NodeType *d_node_types, bool * d_bounce_back_dir, double * d_bounce_back_delta,
                          int nx, int ny, int i, int index)
{
  int forward_index = find_forward_index(index, nx, ny, i, d_coeff);
  double ux_wall = d_ux[forward_index];
  double uy_wall = d_uy[forward_index];
  double rho_wall = d_rho[forward_index];

  if(d_node_types[forward_index] == NodeType::outlet)
  {
    int backward_index = find_backward_index(index, nx, ny, i, d_coeff, d_bb_indexes);
    double ux_fluid = d_ux[backward_index];
    double uy_fluid = d_uy[backward_index];

    ux_wall = (d_ux[forward_index] + ux_fluid) / 2;
    uy_wall = (d_uy[forward_index] + uy_fluid) / 2;
    
    rho_wall = 0.8 * (2 * (d_f_post[index * dir + 1] + d_f_post[index * dir + 5] + d_f_post[index * dir + 8]) + d_f_post[index * dir + 0] + d_f_post[index * dir + 2] + d_f_post[index * dir + 4]) / (1. - ux_wall);
  }

  if(check_backward(index, nx, ny, i, d_coeff, d_bb_indexes, d_node_types))
  {
    double cx = d_coeff[2 * i];
    double cy = d_coeff[2 * i + 1];

    d_f_adj[index * dir + d_bb_indexes[i]] = -d_f_post[index * dir + i] +
                          2 * d_weights[i] * rho_wall *
                          (1 + 4.5 * (cx * ux_wall + cy * uy_wall) * (cx * ux_wall + cy * uy_wall) -
                          3.5 * (ux_wall * ux_wall + uy_wall * uy_wall));
  }
  else
  {
    d_f_adj[index * dir + d_bb_indexes[i]] = d_f_post[index * dir + i];
  }
}

__global__ void collide_and_stream_kernel(
  const int dir, const double *d_weights, const double *d_coeff, const int *d_bb_indexes,
  double *d_f_pre, double *d_f_post, double *d_f_adj, 
  double *d_ux, double *d_uy, double *d_rho, 
  NodeType *d_node_types, bool * d_bounce_back_dir, int nx, int ny, double tau) 
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int n = nx * ny;
  if(index < n) 
  {
    if(d_node_types[index] == NodeType::fluid || d_node_types[index] == NodeType::boundary) 
    {
      // Collision step
      for(int i = 0; i < dir; ++i)
      {
        double feq = compute_equilibrium(d_weights, d_coeff, d_rho[index], d_ux[index], d_uy[index], i);
        d_f_post[index * dir + i] = d_f_pre[index * dir + i] - (d_f_pre[index * dir + i] - feq) / tau;
      }

      // Streaming step
      d_f_adj[index * dir + 0] = d_f_post[index * dir + 0];
      for(int i = 0; i < dir; ++i) 
      {
        if(!d_bounce_back_dir[index * dir + i]) {
          int index_new = find_forward_index(index, nx, ny, i, d_coeff);
          d_f_adj[index_new * dir + i] = d_f_post[index * dir + i];
        }
      }
    }
  }
}

__global__ void apply_BCs_and_compute_quantities_kernel(
  const int dir, const double *d_weights, const double *d_coeff, const int *d_bb_indexes,
  double *d_f_pre, double *d_f_post, double *d_f_adj,
  double *d_ux, double *d_uy, double *d_rho,
  double *d_drag, double *d_lift, bool obstacle_present,
  NodeType *d_node_types, bool * d_bounce_back_dir, double * d_bounce_back_delta, int nx, int ny) 
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int n = nx * ny;
  // printf("index: %d\n", index);
  if(index < n)
  {
    if(d_node_types[index] == NodeType::fluid || d_node_types[index] == NodeType::boundary)
    {
      if(d_node_types[index] == NodeType::boundary)
      {
        // Apply boundary conditions
        for(int i = 0; i < dir; ++i)
        {
          if(d_bounce_back_dir[index * dir + i])
          {
            int index_new = find_forward_index(index, nx, ny, i, d_coeff);

            if(d_node_types[index_new] == NodeType::solid ||
              d_node_types[index_new] == NodeType::obstacle ||
              d_node_types[index_new] == NodeType::inlet)
            {
              // Interpolated Bounce-Back
              apply_IBB(dir, d_weights, d_coeff, d_bb_indexes, d_f_post, d_f_adj, d_ux, d_uy, d_rho, d_node_types, d_bounce_back_dir, d_bounce_back_delta, nx, ny, i, index);
            }
            else if(d_node_types[index_new] == NodeType::outlet)
            {
              // Anti Bounce-Back
              apply_anti_BB(dir, d_weights, d_coeff, d_bb_indexes, d_f_post, d_f_adj, d_ux, d_uy, d_rho, d_node_types, d_bounce_back_dir, d_bounce_back_delta, nx, ny, i, index);
            }
            else
            {
              printf("Error: Invalid BCs type at index %d\n", index);
              return;
            }
          }
        }

        // Compute drag and lift
        if(obstacle_present)
        {
          double dr = 0.0;
          double lf = 0.0;
          for(int i = 0; i < dir; ++i)
          {
            int forward_index = find_forward_index(index, nx, ny, i, d_coeff);
            if(d_node_types[forward_index] == NodeType::obstacle)
            {
              double cx = d_coeff[2 * i];
              double cy = d_coeff[2 * i + 1];
              double cx_bb = d_coeff[2 * d_bb_indexes[i]];
              double cy_bb = d_coeff[2 * d_bb_indexes[i] + 1];

              dr += cx * d_f_pre[index * dir + i] - cx_bb * d_f_adj[index * dir + d_bb_indexes[i]];
              lf -= cy * d_f_pre[index * dir + i] - cy_bb * d_f_adj[index * dir + d_bb_indexes[i]];
            }
          }
          atomicAdd(d_drag, dr);
          atomicAdd(d_lift, lf);
        }
      }

      // Update f
      for(int i = 0; i < dir; ++i)
      {
        d_f_pre[index * dir + i] = d_f_adj[index * dir + i];
      }

      // Compute macroscopic quantities
      double rho = 0.0;
      double ux = 0.0;
      double uy = 0.0;
      for(int i = 0; i < dir; ++i)
      {
        double f = d_f_pre[index * dir + i];
        rho += f;
        ux += f * d_coeff[2 * i];
        uy += f * d_coeff[2 * i + 1];
      }
      d_rho[index] = rho;
      d_ux[index] = ux / rho;
      d_uy[index] = uy / rho;
    }
  }
}

void
lbm_gpu::cuda_simulation(unsigned int nx, 
                        unsigned int ny, 
                        std::vector<Node> &nodes,
                        double tau,
                        double dt,
                        unsigned int save_iter,
                        unsigned int max_iter)
{

  const int n = nx * ny;
  bool obstacle_present = false;

  // Constants for CUDA kernel
  const int dir = Node::dir;
  const double *weights = vectorToArray(Node::weights);
  const double *coeff = vector2DToArray(Node::coeff);
  const int *bb_indexes = vectorToArray(Node::bb_indexes);

  // Copy constant variables to device
  double *d_weights, *d_coeff;
  int *d_bb_indexes;

  CUDA_CHECK(hipMalloc((void **) &d_weights, dir * sizeof(double)));
  CUDA_CHECK(hipMalloc((void **) &d_coeff, 2 * dir * sizeof(double)));
  CUDA_CHECK(hipMalloc((void **) &d_bb_indexes, dir * sizeof(int)));

  CUDA_CHECK(hipMemcpy(d_weights, weights, dir * sizeof(double), hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(d_coeff, coeff, 2 * dir * sizeof(double), hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(d_bb_indexes, bb_indexes, dir * sizeof(int), hipMemcpyHostToDevice));


  std::vector<double> lift_out(max_iter, 0.0);
  std::vector<double> drag_out(max_iter, 0.0);
  
  // Host variables
  double * host_f_pre, * host_f_post, * host_f_adj, * host_ux, * host_uy, * host_rho, * host_drag, * host_lift, * host_bounce_back_delta;
  int * host_coord;
  bool * host_bounce_back_dir;
  NodeType * host_node_types;

  // Device variables
  double * d_f_pre, * d_f_post, * d_f_adj, * d_ux, * d_uy, * d_rho, * d_drag, * d_lift, * d_bounce_back_delta;
  int * d_coord;
  bool * d_bounce_back_dir;
  NodeType * d_node_types;

  // Allocate memory on the host
  CUDA_CHECK(hipHostMalloc((void **) &host_f_pre, n * dir * sizeof(double)));
  CUDA_CHECK(hipHostMalloc((void **) &host_f_post, n * dir * sizeof(double)));
  CUDA_CHECK(hipHostMalloc((void **) &host_f_adj, n * dir * sizeof(double)));
  CUDA_CHECK(hipHostMalloc((void **) &host_ux, n * sizeof(double)));
  CUDA_CHECK(hipHostMalloc((void **) &host_uy, n * sizeof(double)));
  CUDA_CHECK(hipHostMalloc((void **) &host_rho, n * sizeof(double)));
  CUDA_CHECK(hipHostMalloc((void **) &host_drag, sizeof(double)));
  CUDA_CHECK(hipHostMalloc((void **) &host_lift, sizeof(double)));
  CUDA_CHECK(hipHostMalloc((void **) &host_coord, n * 2 * sizeof(int)));
  CUDA_CHECK(hipHostMalloc((void **) &host_bounce_back_delta, n * dir * sizeof(double)));
  CUDA_CHECK(hipHostMalloc((void **) &host_bounce_back_dir, n * dir * sizeof(bool)));
  CUDA_CHECK(hipHostMalloc((void **) &host_node_types, n * sizeof(NodeType)));

  // Allocate memory on the device
  CUDA_CHECK(hipMalloc((void **) &d_f_pre, n * dir * sizeof(double)));
  CUDA_CHECK(hipMalloc((void **) &d_f_post, n * dir * sizeof(double)));
  CUDA_CHECK(hipMalloc((void **) &d_f_adj, n * dir * sizeof(double)));
  CUDA_CHECK(hipMalloc((void **) &d_ux, n * sizeof(double)));
  CUDA_CHECK(hipMalloc((void **) &d_uy, n * sizeof(double)));
  CUDA_CHECK(hipMalloc((void **) &d_rho, n * sizeof(double)));
  CUDA_CHECK(hipMalloc((void **) &d_drag, sizeof(double)));
  CUDA_CHECK(hipMalloc((void **) &d_lift, sizeof(double)));
  CUDA_CHECK(hipMalloc((void **) &d_coord, n * 2 * sizeof(int)));
  CUDA_CHECK(hipMalloc((void **) &d_bounce_back_delta, n * dir * sizeof(double)));
  CUDA_CHECK(hipMalloc((void **) &d_bounce_back_dir, n * dir * sizeof(bool)));
  CUDA_CHECK(hipMalloc((void **) &d_node_types, n * sizeof(NodeType)));

  // Set host data
  //TODO: use openMP?
  #pragma omp parallel for
  for(unsigned int index = 0; index < n; index++)
  {
    std::vector<double> temp_f_pre = nodes[index].get_f_pre();
    std::vector<double> temp_f_post = nodes[index].get_f_post();
    std::vector<double> temp_f_adj = nodes[index].get_f_adj();
    std::vector<double> temp_bounce_back_delta = nodes[index].get_bounce_back_delta();
    std::vector<bool> temp_bounce_back_dir = nodes[index].get_bounce_back_dir();

    host_ux[index] = nodes[index].get_ux();
    host_uy[index] = nodes[index].get_uy();
    host_rho[index] = nodes[index].get_rho();
    host_node_types[index] = nodes[index].get_node_type();
    
    if(host_node_types[index] == NodeType::obstacle && !obstacle_present) {
      std::cout << "Obstacle present\n" << std::endl;
      obstacle_present = true;
    }

    for(unsigned int i = 0; i < dir; i++)
    {
      host_f_pre[index * dir + i] = temp_f_pre[i];
      host_f_post[index * dir + i] = temp_f_post[i];
      host_f_adj[index * dir + i] = temp_f_adj[i];
      host_bounce_back_delta[index * dir + i] = temp_bounce_back_delta[i];
      host_bounce_back_dir[index * dir + i] = temp_bounce_back_dir[i];
    }

    for(unsigned int i = 0; i < 2; i++)
    {
      host_coord[index * 2 + i] = nodes[index].get_coord()[i];
    }
    
  }
  *host_drag = 0.0;
  *host_lift = 0.0;

  std::cout << "Copying data to device\n" << std::endl;
  // Copy data from host to device
  CUDA_CHECK(hipMemcpy(d_f_pre, host_f_pre, n * dir * sizeof(double), hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(d_f_post, host_f_post, n * dir * sizeof(double), hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(d_f_adj, host_f_adj, n * dir * sizeof(double), hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(d_ux, host_ux, n * sizeof(double), hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(d_uy, host_uy, n * sizeof(double), hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(d_rho, host_rho, n * sizeof(double), hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(d_drag, host_drag, sizeof(double), hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(d_lift, host_lift, sizeof(double), hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(d_coord, host_coord, n * 2 * sizeof(int), hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(d_bounce_back_delta, host_bounce_back_delta, n * dir * sizeof(double), hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(d_bounce_back_dir, host_bounce_back_dir, n * dir * sizeof(bool), hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(d_node_types, host_node_types, n * sizeof(NodeType), hipMemcpyHostToDevice));

  // Free host memory
  CUDA_CHECK(hipHostFree(host_f_pre));
  CUDA_CHECK(hipHostFree(host_f_post));
  CUDA_CHECK(hipHostFree(host_f_adj));
  CUDA_CHECK(hipHostFree(host_coord));
  CUDA_CHECK(hipHostFree(host_bounce_back_delta));
  CUDA_CHECK(hipHostFree(host_bounce_back_dir));
  CUDA_CHECK(hipHostFree(host_node_types));

  // Run simulation
  std::cout << "Running simulation\n" << std::endl;
  auto start_time = std::chrono::high_resolution_clock::now();
  unsigned int iter = 0;
  double total_time = 0.0;
  // std::cout << "Create folder and files\n" << std::endl;
  // Delete the output_results directory if it exists
  if (std::filesystem::exists("output_results")) {
    std::filesystem::remove_all("output_results");
    std::filesystem::create_directory("output_results");
  }
  else{
    std::filesystem::create_directory("output_results");
  }

  if (std::filesystem::exists("output_animations")) {
    std::filesystem::remove_all("output_animations");
    std::filesystem::create_directory("output_animations");
  }
  else{
    std::filesystem::create_directory("output_animations");
  }
  
  std::string u_filename = "output_results/velocity_out.txt";
  std::string ux_filename = "output_results/ux_out.txt";
  std::string uy_filename = "output_results/uy_out.txt";
  std::string rho_filename = "output_results/rho_out.txt";

  std::ofstream u_file(u_filename);
  std::ofstream ux_file(ux_filename);
  std::ofstream uy_file(uy_filename);
  std::ofstream rho_file(rho_filename);

  // std::cout << "Save initial conditions\n" << std::endl;
  std::vector<double> vec_ux(nx * ny), vec_uy(nx * ny), vec_rho(nx * ny);
  vec_ux = arrayToVector(host_ux, nx * ny);
  vec_uy = arrayToVector(host_uy, nx * ny);
  vec_rho = arrayToVector(host_rho, nx * ny);
  // Save the initial conditions
  writeResults(u_file, ux_file, uy_file, rho_file, vec_ux, vec_uy, vec_rho, nx, ny);

  iter = iter + 1;

  double lift_array[max_iter + 1] {0.};
  double drag_array[max_iter + 1] {0.};

  std::cout << "Start simulation loop\n" << std::endl;
  while(iter <= max_iter) {
    auto iter_start_time = std::chrono::high_resolution_clock::now();

    if(iter % save_iter == 0 || iter == max_iter - 1) {
      std::cout << "Iteration: " << iter << std::endl;
      std::cout << "Time: " << iter * dt << std::endl;
      std::cout << "Collision and streaming" << std::endl;
    }

    // Define block size (number of threads per block)
    int blockSize = 256; // 256 or 512

    // Calculate grid size (number of blocks per grid)
    int gridSize = (nx * ny + blockSize - 1) / blockSize;

    // Launch CUDA kernel for collision and streaming
    collide_and_stream_kernel<<<gridSize, blockSize>>>(dir, d_weights, d_coeff, d_bb_indexes, 
                                                      d_f_pre, d_f_post, d_f_adj,
                                                      d_ux, d_uy, d_rho, 
                                                      d_node_types, d_bounce_back_dir, nx, ny, tau);
    
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());
    if(iter % save_iter == 0 || iter == max_iter - 1) {
      std::cout << "Physical quantities evaluation\n" << std::endl;
    }

    // Launch CUDA kernel for applying boundary conditions and computing physical quantities
    apply_BCs_and_compute_quantities_kernel<<<gridSize, blockSize>>>(dir, d_weights, d_coeff, d_bb_indexes, 
                                                                    d_f_pre, d_f_post, d_f_adj,
                                                                    d_ux, d_uy, d_rho,
                                                                    d_drag, d_lift, obstacle_present,
                                                                    d_node_types, d_bounce_back_dir, d_bounce_back_delta, nx, ny);

    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());

    // Copy lift and drag results from device to host
    if(obstacle_present)
    {  
      CUDA_CHECK(hipMemcpy(host_lift, d_lift, sizeof(double), hipMemcpyDeviceToHost));
      CUDA_CHECK(hipMemcpy(host_drag, d_drag, sizeof(double), hipMemcpyDeviceToHost));
      
      lift_array[iter] = *host_lift;
      drag_array[iter] = *host_drag;

      *host_lift = 0.0;
      *host_drag = 0.0;
      CUDA_CHECK(hipMemcpy(d_lift, host_lift, sizeof(double), hipMemcpyHostToDevice));
      CUDA_CHECK(hipMemcpy(d_drag, host_drag, sizeof(double), hipMemcpyHostToDevice));
    }

    if(iter % save_iter == 0 || iter == max_iter - 1) {
      // Copy results from device to host
      CUDA_CHECK(hipMemcpy(host_ux, d_ux, n * sizeof(double), hipMemcpyDeviceToHost));
      CUDA_CHECK(hipMemcpy(host_uy, d_uy, n * sizeof(double), hipMemcpyDeviceToHost));
      CUDA_CHECK(hipMemcpy(host_rho, d_rho, n * sizeof(double), hipMemcpyDeviceToHost));

      vec_ux = arrayToVector(host_ux, nx * ny);
      vec_uy = arrayToVector(host_uy, nx * ny);
      vec_rho = arrayToVector(host_rho, nx * ny);

      writeResults(u_file, ux_file, uy_file, rho_file, vec_ux, vec_uy, vec_rho, nx, ny);
    }

    iter = iter + 1;
    auto iter_end_time = std::chrono::high_resolution_clock::now();
    total_time += std::chrono::duration<double>(iter_end_time - iter_start_time).count();
  }

  // Save the lift and drag results
  if(obstacle_present)
  { 
    lift_out = arrayToVector(lift_array, max_iter);
    drag_out = arrayToVector(drag_array, max_iter);
    std::string lift_drag_filename = "output_results/lift_&_drag.txt";
    std::ofstream lift_drag_file(lift_drag_filename);
    lift_drag_file << "Lift:\n";
    // Save the lift and drag
    for(unsigned int t=0; t<max_iter; ++t){
      lift_drag_file << lift_out[t] << " ";
    }
    
    lift_drag_file << "\nDrag:\n";
    // Save the lift and drag
    for(unsigned int t=0; t<max_iter; ++t){
      lift_drag_file << drag_out[t] << " ";
    }
    lift_drag_file.close();
  }

  auto end_time = std::chrono::high_resolution_clock::now();
  std::chrono::duration<double> elapsed_time = end_time - start_time;
  double mean_time_per_iter = total_time / max_iter;
  std::cout << "Simulation completed in " << elapsed_time.count() << " seconds" << std::endl;
  std::cout << "Mean time per iteration: " << mean_time_per_iter << " seconds.\n" << std::endl;

  u_file.close();
  ux_file.close();
  uy_file.close();
  rho_file.close();

  // Free host memory
  CUDA_CHECK(hipHostFree(host_ux));
  CUDA_CHECK(hipHostFree(host_uy));
  CUDA_CHECK(hipHostFree(host_rho));
  CUDA_CHECK(hipHostFree(host_drag));
  CUDA_CHECK(hipHostFree(host_lift));
  
}