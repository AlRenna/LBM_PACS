#include "hip/hip_runtime.h"
/**
 * @file
 *
 * @author Alessandro Renna <alessandro1.renna@mail.polimi.it>
 * @author Mattia Marzotto <mattia.marzotto@mail.polimi.it>
 */

#include "lattice_gpu.cuh"

// __global__ void kernel()
// {
//   printf("Hello from block %d, thread %d\n", blockIdx.x, threadIdx.x);
// }

void
lbm_gpu::cuda_simulation()
{
  std::cout << "Hello from CUDA!" << std::endl;
  // kernel<<<1, 1>>>();
}