#include "hip/hip_runtime.h"
/**
 * @file
 *
 * @author Alessandro Renna <alessandro1.renna@mail.polimi.it>
 * @author Mattia Marzotto <mattia.marzotto@mail.polimi.it>
 */

#include "src/lattice_gpu.cuh"

// __global__ void hello(void)
// {
//   printf("Hello from block %d, thread %d\n", blockIdx.x, threadIdx.x);
// }

#define CUDA_CHECK(call) \
  do { \
    hipError_t err = call; \
    if (err != hipSuccess) { \
      std::cerr << "CUDA error in " << __FILE__ << " at line " << __LINE__ << ": " \
                << hipGetErrorString(err) << std::endl; \
      exit(EXIT_FAILURE); \
    } \
  } while (0)


__device__ double compute_equilibrium(const double *d_weights, const double *d_coeff,
                                      double rho, double ux, double uy, int i)
{
  double weight = d_weights[i];
  double cx = d_coeff[2 * i];
  double cy = d_coeff[2 * i + 1];
  double u_dot_c = ux * cx + uy * cy;
  double u_sq = ux * ux + uy * uy;
  return weight * rho * (1.0 + 3.0 * u_dot_c + 4.5 * u_dot_c * u_dot_c - 1.5 * u_sq);
}

__device__ int find_forward_index(int current_index, int nx, int ny, int i, const double *d_coeff)
{
  int x = current_index % nx;
  int y = current_index / nx;
  double cx = d_coeff[2 * i];
  double cy = d_coeff[2 * i + 1];
  int x_new = x + cx;
  int y_new = y + cy;
  return y_new * nx + x_new;
}

__device__ void apply_IBB(const int dir, const double *d_weights, const double *d_coeff, const int *d_bb_indexes, double current_time,
                          double *d_f_post, double *d_f_adj,
                          double *d_ux, double *d_uy, double *d_rho,
                          NodeType *d_node_types, bool * d_bounce_back_dir, double * d_bounce_back_delta,
                          int nx, int ny, int i, int index)
{
  int forward_index = find_forward_index(index, nx, ny, i, d_coeff);
  double ux_wall = d_ux[forward_index] * (1/ (1 + std::exp(-25 *(current_time - 0.2))));
  double uy_wall = d_uy[forward_index] * (1/ (1 + std::exp(-25 *(current_time - 0.2))));

  // check if the node in the backward direction is a fluid or boundary node
  if(!d_bounce_back_dir[index * dir + d_bb_indexes[i]])
  {
    double cx = d_coeff[2 * i];
    double cy = d_coeff[2 * i + 1];

    double f_adj_post_coll = d_f_adj[index * dir + i];
    d_f_adj[index * dir + d_bb_indexes[i]] = (2 * d_bounce_back_delta[index * dir + i] * d_f_post[index * dir + i] + 
                    (1 - 2 * d_bounce_back_delta[index * dir + i]) * f_adj_post_coll) * 
                    (d_bounce_back_delta[index * dir + i] < 0.5) +
                    (1. / (2 * d_bounce_back_delta[index * dir + i]) * d_f_post[index * dir + i] + 
                    ((2 * d_bounce_back_delta[index * dir + i] - 1.) / (2 * d_bounce_back_delta[index * dir + i])) * d_f_post[index * dir + d_bb_indexes[i]]) *
                    (d_bounce_back_delta[index * dir + i] >= 0.5) - 
                    (ux_wall * cx + uy_wall * cy) * d_weights[i] * 6;
  }
  else
  {
    d_f_adj[index * dir + d_bb_indexes[i]] = d_f_post[index * dir + i];
  }
}

__device__ void apply_BB(const int dir, const double *d_weights, const double *d_coeff, const int *d_bb_indexes, double current_time,
                          double *d_f_post, double *d_f_adj,
                          double *d_ux, double *d_uy, bool * d_bounce_back_dir,
                          int nx, int ny, int i, int index)
{
  int forward_index = find_forward_index(index, nx, ny, i, d_coeff);
  double ux_wall = d_ux[forward_index] * (1/ (1 + std::exp(-25 *(current_time - 0.2))));
  double uy_wall = d_uy[forward_index] * (1/ (1 + std::exp(-25 *(current_time - 0.2))));

  // check if the node in the backward direction is a fluid or boundary node
  if(!d_bounce_back_dir[index * dir + d_bb_indexes[i]])
  {
    double cx = d_coeff[2 * i];
    double cy = d_coeff[2 * i + 1];

    d_f_adj[index * dir + d_bb_indexes[i]] = d_f_post[index * dir + i] -
                    (ux_wall * cx + uy_wall * cy) * d_weights[i] * 6;
  }
  else
  {
    d_f_adj[index * dir + d_bb_indexes[i]] = d_f_post[index * dir + i];
  }
}

__device__ void apply_ZouHe(const int dir, const double *d_coeff, double *d_f_adj,
                           double *d_rho, ZouHeType * d_zou_he_types,
                          int nx, int ny, int index)
{
  double u_wall = 0.0;
  double rho_wall = 0.0;
  int forward_index = 0;

  switch (d_zou_he_types[index])
  {
    case ZouHeType::none:
      // No Zou-He boundary condition
      break;
    case ZouHeType::right:
      forward_index = find_forward_index(index, nx, ny, 1, d_coeff);
      rho_wall = d_rho[forward_index];
      u_wall = d_f_adj[index * dir + 0] + d_f_adj[index * dir + 2] + d_f_adj[index * dir + 4] + 2.0 * (d_f_adj[index * dir + 1] + d_f_adj[index * dir + 5] + d_f_adj[index * dir + 8]) - rho_wall;
      d_f_adj[index * dir + 3] = d_f_adj[index * dir + 1] - 2.0 / 3.0 * u_wall;
      d_f_adj[index * dir + 6] = d_f_adj[index * dir + 8] - 0.5 * (d_f_adj[index * dir + 2] - d_f_adj[index * dir + 4]) - 1.0 / 6.0 * u_wall;
      d_f_adj[index * dir + 7] = d_f_adj[index * dir + 5] + 0.5 * (d_f_adj[index * dir + 2] - d_f_adj[index * dir + 4]) - 1.0 / 6.0 * u_wall;
      break;
    case ZouHeType::top:
      forward_index = find_forward_index(index, nx, ny, 2, d_coeff);
      rho_wall = d_rho[forward_index];
      u_wall = d_f_adj[index * dir + 0] + d_f_adj[index * dir + 1] + d_f_adj[index * dir + 3] + 2.0 * (d_f_adj[index * dir + 2] + d_f_adj[index * dir + 5] + d_f_adj[index * dir + 6]) - rho_wall;
      d_f_adj[index * dir + 4] = d_f_adj[index * dir + 2] - 2.0 / 3.0 * u_wall;
      d_f_adj[index * dir + 7] = d_f_adj[index * dir + 5] - 0.5 * (d_f_adj[index * dir + 3] - d_f_adj[index * dir + 1]) - 1.0 / 6.0 * u_wall;
      d_f_adj[index * dir + 8] = d_f_adj[index * dir + 6] + 0.5 * (d_f_adj[index * dir + 3] - d_f_adj[index * dir + 1]) - 1.0 / 6.0 * u_wall;
      break;
    case ZouHeType::left:
      forward_index = find_forward_index(index, nx, ny, 3, d_coeff);
      rho_wall = d_rho[forward_index];
      u_wall = d_f_adj[index * dir + 0] + d_f_adj[index * dir + 2] + d_f_adj[index * dir + 4] + 2.0 * (d_f_adj[index * dir + 3] + d_f_adj[index * dir + 6] + d_f_adj[index * dir + 7]) - rho_wall;
      d_f_adj[index * dir + 1] = d_f_adj[index * dir + 3] - 2.0 / 3.0 * u_wall;
      d_f_adj[index * dir + 5] = d_f_adj[index * dir + 7] - 0.5 * (d_f_adj[index * dir + 2] - d_f_adj[index * dir + 4]) - 1.0 / 6.0 * u_wall;
      d_f_adj[index * dir + 8] = d_f_adj[index * dir + 6] + 0.5 * (d_f_adj[index * dir + 2] - d_f_adj[index * dir + 4]) - 1.0 / 6.0 * u_wall;
      break;
    case ZouHeType::bottom:
      forward_index = find_forward_index(index, nx, ny, 4, d_coeff);
      rho_wall = d_rho[forward_index];
      u_wall = d_f_adj[index * dir + 0] + d_f_adj[index * dir + 1] + d_f_adj[index * dir + 3] + 2.0 * (d_f_adj[index * dir + 4] + d_f_adj[index * dir + 7] + d_f_adj[index * dir + 8]) - rho_wall;
      d_f_adj[index * dir + 2] = d_f_adj[index * dir + 4] - 2.0 / 3.0 * u_wall;
      d_f_adj[index * dir + 5] = d_f_adj[index * dir + 7] - 0.5 * (d_f_adj[index * dir + 1] - d_f_adj[index * dir + 3]) - 1.0 / 6.0 * u_wall;
      d_f_adj[index * dir + 6] = d_f_adj[index * dir + 8] + 0.5 * (d_f_adj[index * dir + 1] - d_f_adj[index * dir + 3]) - 1.0 / 6.0 * u_wall;
      break;
    default:
      printf("Error: Invalid ZouHeType at index %d\n", index);
      return;
  }

}

__global__ void collide_and_stream_kernel(
  const int dir, const double *d_weights, const double *d_coeff, const int *d_bb_indexes,
  double *d_f_pre, double *d_f_post, double *d_f_adj, 
  double *d_ux, double *d_uy, double *d_rho, 
  NodeType *d_node_types, bool * d_bounce_back_dir, int nx, int ny, double tau) 
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int n = nx * ny;
  if(index < n) 
  {
    if(d_node_types[index] == NodeType::fluid || d_node_types[index] == NodeType::boundary) 
    {
      // Collision step
      for(int i = 0; i < dir; ++i)
      {
        double feq = compute_equilibrium(d_weights, d_coeff, d_rho[index], d_ux[index], d_uy[index], i);
        d_f_post[index * dir + i] = d_f_pre[index * dir + i] - (d_f_pre[index * dir + i] - feq) / tau;
      }

      // Streaming step
      d_f_adj[index * dir + 0] = d_f_post[index * dir + 0];
      for(int i = 0; i < dir; ++i) 
      {
        if(!d_bounce_back_dir[index * dir + i]) {
          int index_new = find_forward_index(index, nx, ny, i, d_coeff);
          d_f_adj[index_new * dir + i] = d_f_post[index * dir + i];
        }
      }
    }
  }
}

__global__ void apply_BCs_and_compute_quantities_kernel(
  const int dir, const double *d_weights, const double *d_coeff, const int *d_bb_indexes, double current_time,
  double *d_f_pre, double *d_f_post, double *d_f_adj,
  double *d_ux, double *d_uy, double *d_rho,
  double *d_drag, double *d_lift, bool obstacle_present,
  NodeType *d_node_types, ZouHeType * d_zou_he_types, bool * d_bounce_back_dir, double * d_bounce_back_delta, int nx, int ny) 
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int n = nx * ny;
  // printf("index: %d\n", index);
  if(index < n)
  {
    if(d_node_types[index] == NodeType::fluid || d_node_types[index] == NodeType::boundary)
    {
      if(d_node_types[index] == NodeType::boundary)
      {
        // Apply boundary conditions
        for(int i = 0; i < dir; ++i)
        {
          if(d_bounce_back_dir[index * dir + i])
          {
            int index_new = find_forward_index(index, nx, ny, i, d_coeff);

            if(d_node_types[index_new] == NodeType::solid ||
              d_node_types[index_new] == NodeType::obstacle ||
              d_node_types[index_new] == NodeType::inlet)
            {
              // Interpolated Bounce-Back
              apply_IBB(dir, d_weights, d_coeff, d_bb_indexes, current_time, d_f_post, d_f_adj, d_ux, d_uy, d_rho, d_node_types, d_bounce_back_dir, d_bounce_back_delta, nx, ny, i, index);
            }
            else if(d_node_types[index_new] == NodeType::outlet)
            {
              continue; // ZouHe is applied after the loop for outlet nodes
            }
            else
            {
              printf("Error: Invalid BCs type at index %d\n", index);
              return;
            }
          }
        }

        apply_ZouHe(dir, d_coeff,  d_f_adj, d_rho,  d_zou_he_types, nx, ny, index);

        // Compute drag and lift
        if(obstacle_present)
        {
          double dr = 0.0;
          double lf = 0.0;
          for(int i = 0; i < dir; ++i)
          {
            int forward_index = find_forward_index(index, nx, ny, i, d_coeff);
            if(d_node_types[forward_index] == NodeType::obstacle)
            {
              double cx = d_coeff[2 * i];
              double cy = d_coeff[2 * i + 1];
              double cx_bb = d_coeff[2 * d_bb_indexes[i]];
              double cy_bb = d_coeff[2 * d_bb_indexes[i] + 1];

              dr += cx * d_f_pre[index * dir + i] - cx_bb * d_f_adj[index * dir + d_bb_indexes[i]];
              lf -= cy * d_f_pre[index * dir + i] - cy_bb * d_f_adj[index * dir + d_bb_indexes[i]];
            }
          }
          atomicAdd(d_drag, dr);
          atomicAdd(d_lift, lf);
        }
      }

      // Update f
      for(int i = 0; i < dir; ++i)
      {
        d_f_pre[index * dir + i] = d_f_adj[index * dir + i];
      }

      // Compute macroscopic quantities
      double rho = 0.0;
      double ux = 0.0;
      double uy = 0.0;
      for(int i = 0; i < dir; ++i)
      {
        double f = d_f_pre[index * dir + i];
        rho += f;
        ux += f * d_coeff[2 * i];
        uy += f * d_coeff[2 * i + 1];
      }
      d_rho[index] = rho;
      d_ux[index] = ux / rho;
      d_uy[index] = uy / rho;
    }
  }
}

void
lbm_gpu::cuda_simulation(unsigned int nx, 
                        unsigned int ny, 
                        std::vector<Node> &nodes,
                        double tau,
                        double dt,
                        double Cx,
                        double Crho,
                        unsigned int save_iter,
                        unsigned int max_iter)
{

  const int n = nx * ny;
  bool obstacle_present = false;

  // Constants for CUDA kernel
  const int dir = Node::dir;
  const double *weights = vectorToArray(Node::weights);
  const double *coeff = vector2DToArray(Node::coeff);
  const int *bb_indexes = vectorToArray(Node::bb_indexes);

  // Copy constant variables to device
  double *d_weights, *d_coeff;
  int *d_bb_indexes;

  CUDA_CHECK(hipMalloc((void **) &d_weights, dir * sizeof(double)));
  CUDA_CHECK(hipMalloc((void **) &d_coeff, 2 * dir * sizeof(double)));
  CUDA_CHECK(hipMalloc((void **) &d_bb_indexes, dir * sizeof(int)));

  CUDA_CHECK(hipMemcpy(d_weights, weights, dir * sizeof(double), hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(d_coeff, coeff, 2 * dir * sizeof(double), hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(d_bb_indexes, bb_indexes, dir * sizeof(int), hipMemcpyHostToDevice));


  std::vector<double> lift_out(max_iter, 0.0);
  std::vector<double> drag_out(max_iter, 0.0);
  
  // Host variables
  double * host_f_pre, * host_f_post, * host_f_adj, * host_ux, * host_uy, * host_rho, * host_drag, * host_lift, * host_bounce_back_delta;
  int * host_coord;
  bool * host_bounce_back_dir;
  NodeType * host_node_types;
  ZouHeType * host_zou_he_types;

  // Device variables
  double * d_f_pre, * d_f_post, * d_f_adj, * d_ux, * d_uy, * d_rho, * d_drag, * d_lift, * d_bounce_back_delta;
  int * d_coord;
  bool * d_bounce_back_dir;
  NodeType * d_node_types;
  ZouHeType * d_zou_he_types;

  // Allocate memory on the host
  CUDA_CHECK(hipHostMalloc((void **) &host_f_pre, n * dir * sizeof(double)));
  CUDA_CHECK(hipHostMalloc((void **) &host_f_post, n * dir * sizeof(double)));
  CUDA_CHECK(hipHostMalloc((void **) &host_f_adj, n * dir * sizeof(double)));
  CUDA_CHECK(hipHostMalloc((void **) &host_ux, n * sizeof(double)));
  CUDA_CHECK(hipHostMalloc((void **) &host_uy, n * sizeof(double)));
  CUDA_CHECK(hipHostMalloc((void **) &host_rho, n * sizeof(double)));
  CUDA_CHECK(hipHostMalloc((void **) &host_drag, sizeof(double)));
  CUDA_CHECK(hipHostMalloc((void **) &host_lift, sizeof(double)));
  CUDA_CHECK(hipHostMalloc((void **) &host_coord, n * 2 * sizeof(int)));
  CUDA_CHECK(hipHostMalloc((void **) &host_bounce_back_delta, n * dir * sizeof(double)));
  CUDA_CHECK(hipHostMalloc((void **) &host_bounce_back_dir, n * dir * sizeof(bool)));
  CUDA_CHECK(hipHostMalloc((void **) &host_node_types, n * sizeof(NodeType)));
  CUDA_CHECK(hipHostMalloc((void **) &host_zou_he_types, n * sizeof(ZouHeType)));

  // Allocate memory on the device
  CUDA_CHECK(hipMalloc((void **) &d_f_pre, n * dir * sizeof(double)));
  CUDA_CHECK(hipMalloc((void **) &d_f_post, n * dir * sizeof(double)));
  CUDA_CHECK(hipMalloc((void **) &d_f_adj, n * dir * sizeof(double)));
  CUDA_CHECK(hipMalloc((void **) &d_ux, n * sizeof(double)));
  CUDA_CHECK(hipMalloc((void **) &d_uy, n * sizeof(double)));
  CUDA_CHECK(hipMalloc((void **) &d_rho, n * sizeof(double)));
  CUDA_CHECK(hipMalloc((void **) &d_drag, sizeof(double)));
  CUDA_CHECK(hipMalloc((void **) &d_lift, sizeof(double)));
  CUDA_CHECK(hipMalloc((void **) &d_coord, n * 2 * sizeof(int)));
  CUDA_CHECK(hipMalloc((void **) &d_bounce_back_delta, n * dir * sizeof(double)));
  CUDA_CHECK(hipMalloc((void **) &d_bounce_back_dir, n * dir * sizeof(bool)));
  CUDA_CHECK(hipMalloc((void **) &d_node_types, n * sizeof(NodeType)));
  CUDA_CHECK(hipMalloc((void **) &d_zou_he_types, n * sizeof(ZouHeType)));

  // Set host data
  #pragma omp parallel for
  for(unsigned int index = 0; index < n; index++)
  {
    std::vector<double> temp_f_pre = nodes[index].get_f_pre();
    std::vector<double> temp_f_post = nodes[index].get_f_post();
    std::vector<double> temp_f_adj = nodes[index].get_f_adj();
    std::vector<double> temp_bounce_back_delta = nodes[index].get_bounce_back_delta();
    std::vector<bool> temp_bounce_back_dir = nodes[index].get_bounce_back_dir();

    host_ux[index] = nodes[index].get_ux();
    host_uy[index] = nodes[index].get_uy();
    host_rho[index] = nodes[index].get_rho();
    host_node_types[index] = nodes[index].get_node_type();
    host_zou_he_types[index] = nodes[index].get_zou_he_type();
    
    if(host_node_types[index] == NodeType::obstacle && !obstacle_present) {
      std::cout << "Obstacle present\n" << std::endl;
      obstacle_present = true;
    }

    for(unsigned int i = 0; i < dir; i++)
    {
      host_f_pre[index * dir + i] = temp_f_pre[i];
      host_f_post[index * dir + i] = temp_f_post[i];
      host_f_adj[index * dir + i] = temp_f_adj[i];
      host_bounce_back_delta[index * dir + i] = temp_bounce_back_delta[i];
      host_bounce_back_dir[index * dir + i] = temp_bounce_back_dir[i];
    }

    for(unsigned int i = 0; i < 2; i++)
    {
      host_coord[index * 2 + i] = nodes[index].get_coord()[i];
    }
    
  }
  *host_drag = 0.0;
  *host_lift = 0.0;

  std::cout << "Copying data to device\n" << std::endl;
  // Copy data from host to device
  CUDA_CHECK(hipMemcpy(d_f_pre, host_f_pre, n * dir * sizeof(double), hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(d_f_post, host_f_post, n * dir * sizeof(double), hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(d_f_adj, host_f_adj, n * dir * sizeof(double), hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(d_ux, host_ux, n * sizeof(double), hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(d_uy, host_uy, n * sizeof(double), hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(d_rho, host_rho, n * sizeof(double), hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(d_drag, host_drag, sizeof(double), hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(d_lift, host_lift, sizeof(double), hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(d_coord, host_coord, n * 2 * sizeof(int), hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(d_bounce_back_delta, host_bounce_back_delta, n * dir * sizeof(double), hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(d_bounce_back_dir, host_bounce_back_dir, n * dir * sizeof(bool), hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(d_node_types, host_node_types, n * sizeof(NodeType), hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(d_zou_he_types, host_zou_he_types, n * sizeof(ZouHeType), hipMemcpyHostToDevice));

  // Free host memory
  CUDA_CHECK(hipHostFree(host_f_pre));
  CUDA_CHECK(hipHostFree(host_f_post));
  CUDA_CHECK(hipHostFree(host_f_adj));
  CUDA_CHECK(hipHostFree(host_coord));
  CUDA_CHECK(hipHostFree(host_bounce_back_delta));
  CUDA_CHECK(hipHostFree(host_bounce_back_dir));
  CUDA_CHECK(hipHostFree(host_node_types));
  CUDA_CHECK(hipHostFree(host_zou_he_types));

  // Run simulation
  std::cout << "Running simulation\n" << std::endl;
  auto start_time = std::chrono::high_resolution_clock::now();
  unsigned int iter = 0;
  double total_time = 0.0;
  // std::cout << "Create folder and files\n" << std::endl;
  // Delete the output_results directory if it exists
  if (std::filesystem::exists("output_results")) {
    std::filesystem::remove_all("output_results");
    std::filesystem::create_directory("output_results");
  }
  else{
    std::filesystem::create_directory("output_results");
  }

  if (std::filesystem::exists("output_animations")) {
    std::filesystem::remove_all("output_animations");
    std::filesystem::create_directory("output_animations");
  }
  else{
    std::filesystem::create_directory("output_animations");
  }
  
  std::string u_filename = "output_results/velocity_out.txt";
  std::string ux_filename = "output_results/ux_out.txt";
  std::string uy_filename = "output_results/uy_out.txt";
  std::string rho_filename = "output_results/rho_out.txt";

  std::ofstream u_file(u_filename);
  std::ofstream ux_file(ux_filename);
  std::ofstream uy_file(uy_filename);
  std::ofstream rho_file(rho_filename);

  // std::cout << "Save initial conditions\n" << std::endl;
  std::vector<double> vec_ux(nx * ny), vec_uy(nx * ny), vec_rho(nx * ny);
  vec_ux = arrayToVector(host_ux, nx * ny);
  vec_uy = arrayToVector(host_uy, nx * ny);
  vec_rho = arrayToVector(host_rho, nx * ny);
  // Save the initial conditions
  writeResults(u_file, ux_file, uy_file, rho_file, vec_ux, vec_uy, vec_rho, nx, ny, Cx, dt, Crho);

  iter = iter + 1;

  double lift_array[max_iter + 1] {0.};
  double drag_array[max_iter + 1] {0.};

  std::cout << "Start simulation loop\n" << std::endl;
  double current_time = 0.0;

  while(iter <= max_iter) {
    auto iter_start_time = std::chrono::high_resolution_clock::now();

    current_time = static_cast<double>(iter) / max_iter;
    
    if(iter % save_iter == 0 || iter == max_iter - 1) {
      std::cout << "Iteration: " << iter << std::endl;
      std::cout << "Time: " << iter * dt << std::endl;
      std::cout << "Collision and streaming" << std::endl;
    }

    // Define block size (number of threads per block)
    int blockSize = 256; // 256 or 512

    // Calculate grid size (number of blocks per grid)
    int gridSize = (nx * ny + blockSize - 1) / blockSize;

    // Launch CUDA kernel for collision and streaming
    collide_and_stream_kernel<<<gridSize, blockSize>>>(dir, d_weights, d_coeff, d_bb_indexes,
                                                      d_f_pre, d_f_post, d_f_adj,
                                                      d_ux, d_uy, d_rho, 
                                                      d_node_types, d_bounce_back_dir, nx, ny, tau);
    
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());
    if(iter % save_iter == 0 || iter == max_iter - 1) {
      std::cout << "Physical quantities evaluation\n" << std::endl;
    }

    // Launch CUDA kernel for applying boundary conditions and computing physical quantities
    apply_BCs_and_compute_quantities_kernel<<<gridSize, blockSize>>>(dir, d_weights, d_coeff, d_bb_indexes, current_time, 
                                                                    d_f_pre, d_f_post, d_f_adj,
                                                                    d_ux, d_uy, d_rho,
                                                                    d_drag, d_lift, obstacle_present,
                                                                    d_node_types, d_zou_he_types, d_bounce_back_dir, d_bounce_back_delta, nx, ny);

    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());

    // Copy lift and drag results from device to host
    if(obstacle_present)
    {  
      CUDA_CHECK(hipMemcpy(host_lift, d_lift, sizeof(double), hipMemcpyDeviceToHost));
      CUDA_CHECK(hipMemcpy(host_drag, d_drag, sizeof(double), hipMemcpyDeviceToHost));
      
      lift_array[iter] = *host_lift;
      drag_array[iter] = *host_drag;

      *host_lift = 0.0;
      *host_drag = 0.0;
      CUDA_CHECK(hipMemcpy(d_lift, host_lift, sizeof(double), hipMemcpyHostToDevice));
      CUDA_CHECK(hipMemcpy(d_drag, host_drag, sizeof(double), hipMemcpyHostToDevice));
    }

    if(iter % save_iter == 0 || iter == max_iter - 1) {
      // Copy results from device to host
      CUDA_CHECK(hipMemcpy(host_ux, d_ux, n * sizeof(double), hipMemcpyDeviceToHost));
      CUDA_CHECK(hipMemcpy(host_uy, d_uy, n * sizeof(double), hipMemcpyDeviceToHost));
      CUDA_CHECK(hipMemcpy(host_rho, d_rho, n * sizeof(double), hipMemcpyDeviceToHost));

      vec_ux = arrayToVector(host_ux, nx * ny);
      vec_uy = arrayToVector(host_uy, nx * ny);
      vec_rho = arrayToVector(host_rho, nx * ny);

      writeResults(u_file, ux_file, uy_file, rho_file, vec_ux, vec_uy, vec_rho, nx, ny, Cx, dt, Crho);
    }

    iter = iter + 1;
    auto iter_end_time = std::chrono::high_resolution_clock::now();
    total_time += std::chrono::duration<double>(iter_end_time - iter_start_time).count();
  }

  // Save the lift and drag results
  if(obstacle_present)
  { 
    // Conversion factor for forces - Cf = Crho * (Cx^4) / (Ct^2)
    double Cf = Crho * (Cx * Cx * Cx * Cx) / (dt * dt);
    lift_out = arrayToVector(lift_array, max_iter);
    drag_out = arrayToVector(drag_array, max_iter);
    std::string lift_drag_filename = "output_results/lift_&_drag.txt";
    std::ofstream lift_drag_file(lift_drag_filename);
    lift_drag_file << "Lift:\n";
    // Save the lift and drag
    for(unsigned int t=0; t<max_iter; ++t){
      lift_drag_file << lift_out[t] * Cf << " ";
    }
    
    lift_drag_file << "\nDrag:\n";
    // Save the lift and drag
    for(unsigned int t=0; t<max_iter; ++t){
      lift_drag_file << drag_out[t] * Cf << " ";
    }
    lift_drag_file.close();
  }

  auto end_time = std::chrono::high_resolution_clock::now();
  std::chrono::duration<double> elapsed_time = end_time - start_time;
  double mean_time_per_iter = total_time / max_iter;
  std::cout << "Simulation completed in " << elapsed_time.count() << " seconds" << std::endl;
  std::cout << "Mean time per iteration: " << mean_time_per_iter << " seconds.\n" << std::endl;

  u_file.close();
  ux_file.close();
  uy_file.close();
  rho_file.close();

  // Free host memory
  CUDA_CHECK(hipHostFree(host_ux));
  CUDA_CHECK(hipHostFree(host_uy));
  CUDA_CHECK(hipHostFree(host_rho));
  CUDA_CHECK(hipHostFree(host_drag));
  CUDA_CHECK(hipHostFree(host_lift));
  
}